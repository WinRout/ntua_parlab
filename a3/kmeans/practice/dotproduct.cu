#include "hip/hip_runtime.h"
// global
__global__
void dot_product(float *out, float *a, float *b, int n){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid>=n) return ;
    float partial_result = a[tid]*b[tid];
    atomicAdd(out, partial_result)
}