#include "hip/hip_runtime.h"

//host code
int main(){
    float a[N], b[N], c[N];
    float *dev_a, *dev_b, *dev_c;

    // Allocate memory on the GPU
    hipMalloc((void**)&dev_a, N*sizeof(float));
    hipMalloc((void**)&dev_b, N*sizeof(float));
    hipMalloc((void**)&dev_c, N*sizeof(float));

    // Initialize data on CPU

    // Copy the array's 'a' and 'b' to the GPU
    hipMemcpy(dev_a, a, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N*sizeof(float), hipMemcpyHostToDevice);

    // Launch the kernel with N/128 thread blocks of 128 threads
    dim1 block(128);
    dim1 grid((N+127)/128);
    vec_add<<grid, block>>(dev_a, dev_b, dev_c, N);
    // Synchronize the device! Kernel Launch is async
    hipDeviceSynchronize();

    // Copy the array 'c' back from the GPU to the CPU
    hipMemcpy(c, dev_c, N*sizeof(float), hipMemcpyDeviceToHost);

    // Free the memory allocated on the GPU
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}

// device code
// version 1
__global__ void vec_add(const float *a, const float *b, float *c, int N){
    int tid = blockDim.x*blockIdx.x+threadIdx.x;
    if (tid >= N)
        return ;
    c[tid] = a[tid] +b[tid];
}

// version 2
__global__ void vec_add_2(const float *a, const float *b, float *c, int N){
    // Grid stride loop
    for (int tid = blockDim.x*blockIdx.x + threadIdx.x; tid < N; tid+=blockDim.x*gridDim.x){
        c[tid] = a[tid] + b[tid];
    }
}
